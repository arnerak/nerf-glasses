#include "hip/hip_runtime.h"
#include "cuda_texture.cuh"

CudaTexture::CudaTexture(size_t width, size_t height, std::vector<uint8_t> bytes, bool bSrgb) {
    const hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();

    CUDA_CHECK(hipMallocArray(&_cuArray, &channelDesc, width, height));
    CUDA_CHECK(hipMemcpy2DToArray(
            _cuArray,
            0,
            0,
            bytes.data(),
            width * sizeof(uchar4),
            width * sizeof(uchar4),
            height,
            hipMemcpyHostToDevice
    ));

    hipResourceDesc resDesc{};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = _cuArray;

    hipTextureDesc texDesc{};
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.readMode = hipReadModeNormalizedFloat; // Convert to floating point so that linear filtering can be used.
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.normalizedCoords = 1;
    texDesc.sRGB = bSrgb ? 1 : 0;

    CUDA_CHECK(hipCreateTextureObject(&_cuTexture, &resDesc, &texDesc, nullptr));
}

CudaTexture::~CudaTexture() {
    hipDestroyTextureObject(_cuTexture);
    hipFreeArray(_cuArray);
}
